#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

#include "lodepng.h"

//each pixel has 4 values:
//Red (R)
//Green (G)
//Blue (B)
//Transparency (T)

//negative filter (reversing the RGB values)
//0-255   max-val    255-40

__global__ void NegativeFilter(unsigned char * inputImage, unsigned char * outputImage){

int r;
int g;
int b;
int t;

int threadIndex = blockDim.x * blockIdx.x + threadIdx.x;

int pixel = threadIndex * 4;

printf("uid = %d\n", pixel);

r = inputImage[pixel];
g = inputImage[pixel+1];
b = inputImage[pixel+2];
t = inputImage[pixel+3];

outputImage[pixel] = 255-r;
outputImage[pixel+1] = 255-g;
outputImage[pixel+2] = 255-b;
outputImage[pixel+3] = t;

}

int main(int argc, char ** argv){

  unsigned int errorDecode; //variable will hold whether there was an issue with loading in the png file
  unsigned char* cpuImage; //this variable will hold all of our image data
  unsigned int width, height; //holds the width and height of image
  
  char * filename = argv[1];
  char * newFilename = argv[2];

  errorDecode = lodepng_decode32_file(&cpuImage, &width, &height, filename); // (where to store the image data, width, height, which file?)
  if(errorDecode){
    printf("error %u: %s\n", errorDecode, lodepng_error_text(errorDecode));
  }
  
  int arraySize = width*height*4;
  int memorySize = arraySize * sizeof(unsigned char);
  
  unsigned char cpuOutImage[arraySize];
  
  unsigned char* gpuInput;
  unsigned char* gpuOutput;
  
  hipMalloc( (void**) &gpuInput, memorySize);
  hipMalloc( (void**) &gpuOutput, memorySize);
  
  hipMemcpy(gpuInput, cpuImage, memorySize, hipMemcpyHostToDevice);
  
  NegativeFilter<<< 1000, 1000 >>>(gpuInput, gpuOutput);
  hipDeviceSynchronize();

  hipMemcpy(cpuOutImage, gpuOutput, memorySize, hipMemcpyDeviceToHost);
  
  unsigned int errorEncode = lodepng_encode32_file(newFilename, cpuOutImage, width, height);
  if(errorEncode) {
  printf("error %u: %s\n", errorEncode, lodepng_error_text(errorEncode));
  }

  //free(image);

}
